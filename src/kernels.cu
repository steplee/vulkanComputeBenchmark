#include "hip/hip_runtime.h"
#include "timer.hpp"

//#define CU_GRP_SZ 16
#define CU_GRP_SZ 8

static __device__ int clamp(int x, int a, int b) {
	return x < a ? a : x > b ? b : x;
}

// Optimized version would
//     1) Fetch global memory to shared memory.
//     2) Use two passes.
//     3) Use device-dependent row pitch.
static __global__ void cuda_naiveGaussianBlur_5(
		float* out, const float* in,
		int W, int H, int C) {

	int y = blockIdx.x * blockDim.x + threadIdx.x;
	int x = blockIdx.y * blockDim.y + threadIdx.y;
	int c = threadIdx.z;

	if (y>=H or x>=W) return;

	// x = np.exp(-np.linalg.norm(np.stack(np.meshgrid(*(np.linspace(-1,1,5),)*2),-1), axis=-1)**2 / 1.); x = x / x.sum()
	const float K[25] = {
		0.0124776415432326, 0.026415167354310425, 0.033917746268994874, 0.026415167354310425, 0.0124776415432326, 
		0.026415167354310425, 0.05592090972790175, 0.07180386941492664, 0.05592090972790175, 0.026415167354310425, 
		0.033917746268994874, 0.07180386941492664, 0.09219799334529334, 0.07180386941492664, 0.033917746268994874, 
		0.026415167354310425, 0.05592090972790175, 0.07180386941492664, 0.05592090972790175, 0.026415167354310425, 
		0.0124776415432326, 0.026415167354310425, 0.033917746268994874, 0.026415167354310425, 0.0124776415432326
	};


	float val = 0.f;

	int k = 0;
	for (int j=-2; j<3; j++)
	for (int i=-2; i<3; i++) {
		int yy = j+y, xx = i+x;
		yy = clamp(yy, 0, H-1);
		xx = clamp(xx, 0, W-1);
		val += in[yy*W*C+xx*C+c] * K[k++];
	}

	out[y*W*C+x*C+c] = val;
}


void run_cuda_naiveGaussianBlur_5(
		float* out, const float* in,
		int W, int H, int C) {

	dim3 blk ( (H+CU_GRP_SZ-1)/CU_GRP_SZ, (W+CU_GRP_SZ-1)/CU_GRP_SZ, 1 );
	dim3 thr ( CU_GRP_SZ, CU_GRP_SZ, C );
	cuda_naiveGaussianBlur_5<<<blk,thr>>>(out,in,W,H,C);
}


void run_cuda_1(Timer& t, int N,
		int W, int H, int C, float* outHost, const float* inHost) {
	float *in, *out;
	hipMalloc(&in, sizeof(float)*W*H*C);
	hipMalloc(&out, sizeof(float)*W*H*C);

	hipMemcpy(in, inHost, 4*H*W*C, hipMemcpyHostToDevice);

	{
		TimerMeasurement<> tm(t,N);
		for (int i=0; i<N; i++) {
			run_cuda_naiveGaussianBlur_5(out,in,W,H,C);
			hipDeviceSynchronize();
		}
		hipDeviceSynchronize();
		hipMemcpy(outHost, out, 4*H*W*C, hipMemcpyDeviceToHost);
	}


	hipFree(in);
	hipFree(out);
}
